#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "GpuPoissonSolver.h"

__global__ 
void jacobiIter(char* matrix, int* col_idx, int* row_ptr, float* unk_vect, float* rhs_vect, int matrix_dim, int matrix_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx >= matrix_dim) return;

	int n = idx + 1 < matrix_dim ? row_ptr[idx + 1] : matrix_size;

	float sum = 0.f;
	for (int i = row_ptr[idx]; i < n; i++) {
		int j = col_idx[i];
		if (idx != j) {
			sum += matrix[i] * unk_vect[j];
		}
	}

	unk_vect[idx] = (rhs_vect[idx] - sum) / -4.f;
}

__global__
void fitRange(float* red, float* green, float* blue, int len)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx >= len) return;

	if (red[idx] < 0.f)     red[idx] = 0.f;
	if (red[idx] > 255.f)   red[idx] = 255.f;

	if (green[idx] < 0.f)   green[idx] = 0.f;
	if (green[idx] > 255.f) green[idx] = 255.f;

	if (blue[idx] < 0.f)    blue[idx] = 0.f;
	if (blue[idx] > 255.f)  blue[idx] = 255.f;
}

void gpuJacobiSolver(char* h_matrix, 
					 int* h_col_idx, 
					 int* h_row_ptr, 
					 float* h_unk_vect_red, 
					 float* h_unk_vect_green, 
					 float* h_unk_vect_blue, 
					 float* h_rhs_vect_red, 
					 float* h_rhs_vect_green,
					 float* h_rhs_vect_blue,
					 int matrix_dim, 
					 int matrix_size, 
					 int iters)
{
	char* d_matrix;
	int* d_col_idx;
	int* d_row_ptr;

	float* d_unk_vect_red;
	float* d_unk_vect_green;
	float* d_unk_vect_blue;

	float* d_rhs_vect_red;
	float* d_rhs_vect_green;
	float* d_rhs_vect_blue;


	hipMalloc((void**) &d_matrix, matrix_size * sizeof(char));
	hipMalloc((void**) &d_col_idx, matrix_size * sizeof(int));
	hipMalloc((void**) &d_row_ptr, matrix_dim * sizeof(int));

	hipMalloc((void**) &d_unk_vect_red, matrix_dim * sizeof(float));
	hipMalloc((void**) &d_unk_vect_green, matrix_dim * sizeof(float));
	hipMalloc((void**) &d_unk_vect_blue, matrix_dim * sizeof(float));

	hipMalloc((void**) &d_rhs_vect_red, matrix_dim * sizeof(float));
	hipMalloc((void**) &d_rhs_vect_green, matrix_dim * sizeof(float));
	hipMalloc((void**) &d_rhs_vect_blue, matrix_dim * sizeof(float));


	hipMemcpy(d_matrix, h_matrix, matrix_size * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_col_idx, h_col_idx, matrix_size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row_ptr, h_row_ptr, matrix_dim * sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_unk_vect_red, h_unk_vect_red, matrix_dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_unk_vect_green, h_unk_vect_green, matrix_dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_unk_vect_blue, h_unk_vect_blue, matrix_dim * sizeof(float), hipMemcpyHostToDevice);
	
	hipMemcpy(d_rhs_vect_red, h_rhs_vect_red, matrix_dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_rhs_vect_green, h_rhs_vect_green, matrix_dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_rhs_vect_blue, h_rhs_vect_blue, matrix_dim * sizeof(float), hipMemcpyHostToDevice);

	int threadsPerBlock = 128;
	int blocks = (matrix_dim / threadsPerBlock) + !!(matrix_dim % threadsPerBlock);
	
	for (int i = 0; i < iters; i++) {
		jacobiIter<<<blocks, threadsPerBlock>>>(d_matrix, d_col_idx, d_row_ptr, d_unk_vect_red, d_rhs_vect_red, matrix_dim, matrix_size);
		jacobiIter<<<blocks, threadsPerBlock>>>(d_matrix, d_col_idx, d_row_ptr, d_unk_vect_green, d_rhs_vect_green, matrix_dim, matrix_size);
		jacobiIter<<<blocks, threadsPerBlock>>>(d_matrix, d_col_idx, d_row_ptr, d_unk_vect_blue, d_rhs_vect_blue, matrix_dim, matrix_size);
	}

	fitRange<<<blocks, threadsPerBlock>>>(d_unk_vect_red, d_unk_vect_green, d_unk_vect_blue, matrix_dim);

	hipMemcpy(h_unk_vect_red, d_unk_vect_red, matrix_dim * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_unk_vect_green, d_unk_vect_green, matrix_dim * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_unk_vect_blue, d_unk_vect_blue, matrix_dim * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_matrix);
	hipFree(d_col_idx);
	hipFree(d_row_ptr);
	hipFree(d_unk_vect_red);
	hipFree(d_unk_vect_green);
	hipFree(d_unk_vect_blue);
	hipFree(d_rhs_vect_red);
	hipFree(d_rhs_vect_green);
	hipFree(d_rhs_vect_blue);
}